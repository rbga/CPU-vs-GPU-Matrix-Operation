﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Function to generate a random positive integer matrix
void generateRandomMatrix(int rows, int cols, int* matrix)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            matrix[i * cols + j] = rand() % 100 + 1;  // Generate random positive integer between 1 and 100
        }
    }
}

// Function to print a matrix
void printMatrix(int rows, int cols, int* matrix)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d\t", matrix[i * cols + j]);
        }
        printf("\n");
    }
    //printf("\n");
}

// Matrix addition using CPU
void matrixAdditionCPU(int rows, int cols, int* matrixA, int* matrixB, int* result)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            result[i * cols + j] = matrixA[i * cols + j] + matrixB[i * cols + j];
        }
    }
}

// Matrix subtraction using CPU
void matrixSubtractionCPU(int rows, int cols, int* matrixA, int* matrixB, int* result)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            result[i * cols + j] = matrixA[i * cols + j] - matrixB[i * cols + j];
        }
    }
}

// Matrix multiplication using CPU
void matrixMultiplicationCPU(int rowsA, int colsA, int colsB, int* matrixA, int* matrixB, int* result)
{
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < colsB; j++)
        {
            int sum = 0;
            for (int k = 0; k < colsA; k++)
            {
                sum += matrixA[i * colsA + k] * matrixB[k * colsB + j];
            }
            result[i * colsB + j] = sum;
        }
    }
}

// Matrix transpose using CPU
void matrixTransposeCPU(int rows, int cols, int* matrix, int* result)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            result[j * rows + i] = matrix[i * cols + j];
        }
    }
}

// Matrix addition using CUDA
__global__ void matrixAdditionGPU(int rows, int cols, int* matrixA, int* matrixB, int* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols)
    {
        result[i * cols + j] = matrixA[i * cols + j] + matrixB[i * cols + j];
    }
}

// Matrix subtraction using CUDA
__global__ void matrixSubtractionGPU(int rows, int cols, int* matrixA, int* matrixB, int* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols)
    {
        result[i * cols + j] = matrixA[i * cols + j] - matrixB[i * cols + j];
    }
}

// Matrix multiplication using CUDA
__global__ void matrixMultiplicationGPU(int rowsA, int colsA, int colsB, int* matrixA, int* matrixB, int* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rowsA && j < colsB)
    {
        int sum = 0;
        for (int k = 0; k < colsA; k++)
        {
            sum += matrixA[i * colsA + k] * matrixB[k * colsB + j];
        }
        result[i * colsB + j] = sum;
    }
}

// Matrix transpose using CUDA
__global__ void matrixTransposeGPU(int rows, int cols, int* matrix, int* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols)
    {
        result[j * rows + i] = matrix[i * cols + j];
    }
}

int main()
{
    int rows, cols;
    printf("Rows and Column must be equal, Enter the number of rows: ");
    scanf("%d", &rows);
    printf("Enter the number of columns: ");
    scanf("%d", &cols);

    if (rows != cols)
    {
        while (rows != cols)
        {
            printf("\n\nError! Rows and Columns must be equal");
            printf("\nEnter the number of rows: ");
            scanf("%d", &rows);
            printf("Enter the number of columns: ");
            scanf("%d", &cols);
            if (rows == cols)
                break;
        }
    }

    int size = rows * cols;
    size_t bytes = size * sizeof(int);

    // Allocate memory for matrices on CPU
    int* matrixA_CPU = (int*)malloc(bytes);
    int* matrixB_CPU = (int*)malloc(bytes);
    int* result_CPU = (int*)malloc(bytes);

    // Allocate memory for matrices on GPU
    int* matrixA_GPU;
    int* matrixB_GPU;
    int* result_GPU;
    hipMalloc((void**)&matrixA_GPU, bytes);
    hipMalloc((void**)&matrixB_GPU, bytes);
    hipMalloc((void**)&result_GPU, bytes);

    // Generate random matrices
    generateRandomMatrix(rows, cols, matrixA_CPU);
    generateRandomMatrix(rows, cols, matrixB_CPU);

    // Print the matrices
    printf("\nMatrix A:\n");
    printMatrix(rows, cols, matrixA_CPU);
    printf("\nMatrix B:\n");
    printMatrix(rows, cols, matrixB_CPU);

    printf("\n------------------------------------------------------------------------\n");

    // Copy matrices from CPU to GPU
    hipMemcpy(matrixA_GPU, matrixA_CPU, bytes, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_GPU, matrixB_CPU, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Perform matrix operations using CPU
    clock_t start_CPU, end_CPU;
    clock_t start_GPU, end_GPU;

    {
        // Matrix Addition - CPU
        start_CPU = clock();
        matrixAdditionCPU(rows, cols, matrixA_CPU, matrixB_CPU, result_CPU);
        end_CPU = clock();
        double cpuTime_Addition = ((double)(end_CPU - start_CPU)) / CLOCKS_PER_SEC;

        printf("\n\nMatrix Addition (CPU):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (CPU): %.6f seconds\n", cpuTime_Addition);
    }
    {
        // Matrix Addition - GPU (CUDA)
        start_GPU = clock();
        matrixAdditionGPU <<<numBlocks, threadsPerBlock >>> (rows, cols, matrixA_GPU, matrixB_GPU, result_GPU);
        hipDeviceSynchronize();
        end_GPU = clock();
        double gpuTime_Addition = ((double)(end_GPU - start_GPU)) / CLOCKS_PER_SEC;

        // Copy the result matrix from GPU to CPU
        hipMemcpy(result_CPU, result_GPU, bytes, hipMemcpyDeviceToHost);

        printf("\nMatrix Addition (GPU - CUDA):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (GPU - CUDA): %.6f seconds\n", gpuTime_Addition);
    }
    
    printf("\n------------------------------------------------------------------------\n");

    {
        // Matrix Subtraction - CPU
        start_CPU = clock();
        matrixSubtractionCPU(rows, cols, matrixA_CPU, matrixB_CPU, result_CPU);
        end_CPU = clock();
        double cpuTime_Subtraction = ((double)(end_CPU - start_CPU)) / CLOCKS_PER_SEC;

        printf("\n\nMatrix Subtraction (CPU):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (CPU): %.6f seconds\n", cpuTime_Subtraction);
    }
    {
        // Matrix Subtraction - GPU (CUDA)
        start_GPU = clock();
        matrixSubtractionGPU <<<numBlocks, threadsPerBlock >>> (rows, cols, matrixA_GPU, matrixB_GPU, result_GPU);
        hipDeviceSynchronize();
        end_GPU = clock();
        double gpuTime_Subtraction = ((double)(end_GPU - start_GPU)) / CLOCKS_PER_SEC;

        // Copy the result matrix from GPU to CPU
        hipMemcpy(result_CPU, result_GPU, bytes, hipMemcpyDeviceToHost);

        printf("\nMatrix Subtraction (GPU - CUDA):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (GPU - CUDA): %.6f seconds\n", gpuTime_Subtraction);
    }

    printf("\n------------------------------------------------------------------------\n");

    {
        // Matrix Multiplication - CPU
        start_CPU = clock();
        matrixMultiplicationCPU(rows, cols, cols, matrixA_CPU, matrixB_CPU, result_CPU);
        end_CPU = clock();
        double cpuTime_Multiplication = ((double)(end_CPU - start_CPU)) / CLOCKS_PER_SEC;

        printf("\n\nMatrix Multiplication (CPU):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (CPU): %.6f seconds\n", cpuTime_Multiplication);
    }
    {
        // Matrix Multiplication - GPU (CUDA)
        start_GPU = clock();
        matrixMultiplicationGPU <<<numBlocks, threadsPerBlock >>> (rows, cols, cols, matrixA_GPU, matrixB_GPU, result_GPU);
        hipDeviceSynchronize();
        end_GPU = clock();
        double gpuTime_Multiplication = ((double)(end_GPU - start_GPU)) / CLOCKS_PER_SEC;

        // Copy the result matrix from GPU to CPU
        hipMemcpy(result_CPU, result_GPU, bytes, hipMemcpyDeviceToHost);

        printf("\nMatrix Multiplication (GPU - CUDA):\n");
        printMatrix(rows, cols, result_CPU);
        printf("Time taken (GPU - CUDA): %.6f seconds\n", gpuTime_Multiplication);
    }

    printf("\n------------------------------------------------------------------------\n");

    {
        // Matrix Transpose - CPU
        start_CPU = clock();
        matrixTransposeCPU(rows, cols, matrixA_CPU, result_CPU);
        end_CPU = clock();
        double cpuTime_Transpose = ((double)(end_CPU - start_CPU)) / CLOCKS_PER_SEC;

        printf("\n\nMatrix Transpose (CPU):\n");
        printMatrix(cols, rows, result_CPU);
        printf("Time taken (CPU): %.6f seconds\n", cpuTime_Transpose);
    }
    {
        // Matrix Transpose - GPU (CUDA)
        start_GPU = clock();
        matrixTransposeGPU <<<numBlocks, threadsPerBlock >>> (rows, cols, matrixA_GPU, result_GPU);
        hipDeviceSynchronize();
        end_GPU = clock();
        double gpuTime_Transpose = ((double)(end_GPU - start_GPU)) / CLOCKS_PER_SEC;

        // Copy the result matrix from GPU to CPU
        hipMemcpy(result_CPU, result_GPU, bytes, hipMemcpyDeviceToHost);

        printf("\nMatrix Transpose (GPU - CUDA):\n");
        printMatrix(cols, rows, result_CPU);
        printf("Time taken (GPU - CUDA): %.6f seconds\n", gpuTime_Transpose);
    }

    printf("\n------------------------------------------------------------------------\n");
    
    // Free memory
    free(matrixA_CPU);
    free(matrixB_CPU);
    free(result_CPU);
    hipFree(matrixA_GPU);
    hipFree(matrixB_GPU);
    hipFree(result_GPU);

    return 0;
}
